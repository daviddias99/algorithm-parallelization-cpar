
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <iostream>

using namespace std;

#define TILE_WIDTH 16

__global__ void MatrixMulKernel(double *Md, double *Nd, double *Pd, int width) {
  __shared__ double Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ double Nds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // Identify the row and column of the Pd element to work on
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  double Pvalue = 0;
  // Loop over the Md and Nd tiles required to compute the Pd element
  for (int m = 0; m < width / TILE_WIDTH; ++m) {
    // Coolaborative loading of Md and Nd tiles into shared memory
    Mds[tx][ty] = Md[(m * TILE_WIDTH + tx) * width + Row];
    Nds[tx][ty] = Nd[Col * width + (m * TILE_WIDTH + ty)];
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) Pvalue += Mds[tx][k] * Nds[k][ty];
    __syncthreads();
  }
}



int main(int argc, char *argv[]) {

  // Get arguments
  int matrixSize = atoi(argv[1]);
  int runs = atoi(argv[2]);

  // allocate and initialize host (CPU) memory
  double *M = (double *)malloc(matrixSize * matrixSize * sizeof(double));
  double *N = (double *)malloc(matrixSize * matrixSize * sizeof(double));
  double *P = (double *)malloc(matrixSize * matrixSize * sizeof(double));

  for (int i = 0; i < matrixSize; i++){
    for (int j = 0; j < matrixSize; j++){
      M[i * matrixSize + j] = (double)1.0;
      N[i * matrixSize + j] = (double)(i + 1);
    }
  }
  // allocate device (GPU) memory
  double *Md, *Nd, *Pd;
  hipMalloc((void **)&Md, matrixSize * matrixSize * sizeof(double));
  hipMalloc((void **)&Nd, matrixSize * matrixSize * sizeof(double));
  hipMalloc((void **)&Pd, matrixSize * matrixSize * sizeof(double));
  // copy host memory to device
  hipMemcpy(Md, M, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Nd, N, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Pd, P, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  dim3 dimGrid(matrixSize/TILE_WIDTH, matrixSize/TILE_WIDTH);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH); 

  for(int i = 0; i < runs; i++) {
    // Start counting
    auto begin = std::chrono::high_resolution_clock::now();
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, matrixSize);

    auto end = chrono::high_resolution_clock::now();
    auto elapsed = chrono::duration_cast<chrono::microseconds>(end - begin);
    cout << 1 << " " << matrixSize << " " << TILE_WIDTH << " " << elapsed.count()/ 1000000.0 << endl;
  }
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);
}