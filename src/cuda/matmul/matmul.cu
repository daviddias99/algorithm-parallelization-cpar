#include "hip/hip_runtime.h"
#include <math.h>
#include <chrono>
#include <iostream>

using namespace std;

#define TILE_WIDTH 32
#define TEST_MODE = true

__global__ void MatrixMulKernel(double *Md, double *Nd, double *Pd, int width) {
  __shared__ double Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ double Nds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // Identify the row and column of the Pd element to work on
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  double Pvalue = 0;
  // Loop over the Md and Nd tiles required to compute the Pd element
  for (int m = 0; m < width / TILE_WIDTH; ++m) {
    // Coolaborative loading of Md and Nd tiles into shared memory
    Mds[tx][ty] = Md[(m * TILE_WIDTH + tx) * width + Row];
    Nds[tx][ty] = Nd[Col * width + (m * TILE_WIDTH + ty)];
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) Pvalue += Mds[tx][k] * Nds[k][ty];
    __syncthreads();
  }

  Pd[Row*width+Col] = Pvalue; 
}

__global__ void MatrixMulKernelBlock(double* Md, double* Nd, double* Pd, int Width)
{
  // Calculate the row index of the Pd element and M
  int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
  // Calculate the column idenx of Pd and N
  int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;
  double Pvalue = 0;
  // each thread computes one element of the block sub-matrix
  for (int k = 0; k < Width; ++k)
    Pvalue += Md[Row * Width + k] * Nd[k * Width + Col];
  Pd[Row * Width + Col] = Pvalue;
} 

int main(int argc, char *argv[]) {

  // Get arguments
  int matrixSize = atoi(argv[1]);
  int runs = atoi(argv[2]);

  // allocate and initialize host (CPU) memory
  double *M = (double *)malloc(matrixSize * matrixSize * sizeof(double));
  double *N = (double *)malloc(matrixSize * matrixSize * sizeof(double));
  double *P = (double *)malloc(matrixSize * matrixSize * sizeof(double));

  for (int i = 0; i < matrixSize; i++){
    for (int j = 0; j < matrixSize; j++){
      M[i * matrixSize + j] = (double)1.0;
      N[i * matrixSize + j] = (double)(i + 1);
    }
  }
  // allocate device (GPU) memory
  double *Md, *Nd, *Pd;
  hipMalloc((void **)&Md, matrixSize * matrixSize * sizeof(double));
  hipMalloc((void **)&Nd, matrixSize * matrixSize * sizeof(double));
  hipMalloc((void **)&Pd, matrixSize * matrixSize * sizeof(double));
  // copy host memory to device
  hipMemcpy(Md, M, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Nd, N, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Pd, P, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  dim3 dimGrid(matrixSize/TILE_WIDTH, matrixSize/TILE_WIDTH);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH); 

  for(int i = 0; i < runs; i++) {
    // Start counting
    auto begin = std::chrono::steady_clock::now();
    MatrixMulKernelBlock<<<dimGrid, dimBlock>>>(Md, Nd, Pd, matrixSize);
    hipDeviceSynchronize(); 
    auto end = std::chrono::steady_clock::now();
    auto elapsed = chrono::duration_cast<std::chrono::microseconds>(end - begin);
    cout << 1 << " " << matrixSize << " " << TILE_WIDTH << " " << elapsed.count()/ 1000000.0 << endl;
    
    if(TEST_MODE){
      float flops =(2.0f * matrixSize * matrixSize * matrixSize / (elapsed.count() / 1000000.0f)) * 1.0e-9f;
      cout << flops << endl;
    }
  }

  hipMemcpy(P, Pd, matrixSize*matrixSize * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);
}