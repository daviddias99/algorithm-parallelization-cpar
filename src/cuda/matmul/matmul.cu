#include "hip/hip_runtime.h"
#include <math.h>
#include <chrono>
#include <iostream>

using namespace std;

#define TEST_MODE false

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
if (code != hipSuccess)
{
fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
if (abort) exit(code);
}
}
#define TILE_WIDTH 32
__global__ void MatrixMulKernelBlockLocalMemFixed(double *Md, double *Nd, double *Pd, int width) {
  __shared__ double Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ double Nds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // Identify the row and column of the Pd element to work on
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  double Pvalue = 0;
  // Loop over the Md and Nd tiles required to compute the Pd element
  for (int m = 0; m < width / TILE_WIDTH; ++m) {
    // Coolaborative loading of Md and Nd tiles into shared memory
    Mds[ty][tx] = Md[Row*width + (m*TILE_WIDTH + tx)];
    Nds[ty][tx] = Nd[Col + (m*TILE_WIDTH + ty)*width];
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) Pvalue += Mds[ty][k] * Nds[k][tx];
    __syncthreads();
  }

  Pd[Row*width+Col] = Pvalue; 
}

__global__ void MatrixMulKernelBlockLocalMem(double *Md, double *Nd, double *Pd, int width, int blockSize) {
  __shared__ double* Mds;
  __shared__ double* Nds;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // Identify the row and column of the Pd element to work on
  int Row = by * blockSize + ty;
  int Col = bx * blockSize + tx;
  double Pvalue = 0;

  if (tx == 0 && ty == 0) {
    Mds = (double*) malloc(blockSize * blockSize * sizeof(double));
    Nds = (double*) malloc(blockSize * blockSize * sizeof(double));
  }
  __syncthreads();
  // Loop over the Md and Nd tiles required to compute the Pd element
  for (int m = 0; m < width / blockSize; ++m) {
    // Coolaborative loading of Md and Nd tiles into shared memory
    Mds[ty*blockSize + tx] = Md[Row*width + (m*blockSize + tx)];
    Nds[ty*blockSize + tx] = Nd[Col + (m*blockSize + ty)*width];
    __syncthreads();
    for (int k = 0; k < blockSize; ++k) Pvalue +=  Mds[ty*blockSize + k] * Nds[k*blockSize + tx];
    __syncthreads();
  }
  __syncthreads();
  // Only one thread may free the memory!
  if (tx == 0 && ty == 0) {
    free(Mds);
    free(Nds);
  }
  Pd[Row*width+Col] = Pvalue; 
}
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

__global__ void MatrixMulKernelBlock(double* Md, double* Nd, double* Pd, int Width, int blockSize)
{
  // Calculate the row index of the Pd element and M
  int Row = blockIdx.y*blockSize + threadIdx.y;
  // Calculate the column idenx of Pd and N
  int Col = blockIdx.x*blockSize + threadIdx.x;
  double Pvalue = 0;
  // each thread computes one element of the block sub-matrix
  for (int k = 0; k < Width; ++k)
    Pvalue += Md[Row * Width + k] * Nd[k * Width + Col];
  Pd[Row * Width + Col] = Pvalue;
} 

int main(int argc, char *argv[]) {

  // Get arguments
  int matrixSize = atoi(argv[1]);
  int op = atoi(argv[2]);
  int runs = atoi(argv[3]);
  int blockSize = atoi(argv[4]);

  // allocate and initialize host (CPU) memory
  double *M = (double *)malloc(matrixSize * matrixSize * sizeof(double));
  double *N = (double *)malloc(matrixSize * matrixSize * sizeof(double));
  double *P = (double *)malloc(matrixSize * matrixSize * sizeof(double));

  for (int i = 0; i < matrixSize; i++){
    for (int j = 0; j < matrixSize; j++){
      M[i * matrixSize + j] = (double)1.0;
      N[i * matrixSize + j] = (double)(i + 1);
    }
  }
  // allocate device (GPU) memory
  double *Md, *Nd, *Pd;
  hipMalloc((void **)&Md, matrixSize * matrixSize * sizeof(double));
  hipMalloc((void **)&Nd, matrixSize * matrixSize * sizeof(double));
  hipMalloc((void **)&Pd, matrixSize * matrixSize * sizeof(double));
  // copy host memory to device
  hipMemcpy(Md, M, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Nd, N, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Pd, P, matrixSize*matrixSize * sizeof(double), hipMemcpyHostToDevice);
  dim3 dimGrid(matrixSize/blockSize, matrixSize/blockSize);
  dim3 dimBlock(blockSize, blockSize); 

  if(op == 2)
    hipDeviceSetLimit(hipLimitMallocHeapSize,matrixSize * matrixSize * sizeof(double));

  for(int i = 0; i < runs; i++) {
    // Start counting
    auto begin = std::chrono::steady_clock::now();

    switch(op) {
      case 1:
        MatrixMulKernelBlockLocalMemFixed<<<dimGrid, dimBlock>>>(Md, Nd, Pd, matrixSize);
        break;
      case 2:
        MatrixMulKernelBlockLocalMem<<<dimGrid, dimBlock>>>(Md, Nd, Pd, matrixSize, blockSize);
        break;
      case 3:
        MatrixMulKernelBlock<<<dimGrid, dimBlock>>>(Md, Nd, Pd, matrixSize, blockSize);
        break;
    }

    hipDeviceSynchronize(); 

    auto end = std::chrono::steady_clock::now();
    auto elapsed = chrono::duration_cast<std::chrono::microseconds>(end - begin);
    cout << 1 << " " << matrixSize << " " << blockSize << " " << elapsed.count()/ 1000000.0  << " N/A" << endl;
    
    if(TEST_MODE){
      gpuErrchk( hipPeekAtLastError() );
      hipMemcpy(P, Pd, matrixSize*matrixSize * sizeof(double), hipMemcpyDeviceToHost);
      cout << P[(matrixSize*(matrixSize-1) + matrixSize - 1)] << endl;
      float flops =(2.0f * matrixSize * matrixSize * matrixSize / (elapsed.count() / 1000000.0f)) * 1.0e-9f;
      cout << flops << endl;
    }
  }


  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);
}